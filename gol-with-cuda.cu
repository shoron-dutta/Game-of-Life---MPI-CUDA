#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<unistd.h>
#include<stdbool.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


// Result from last compute of world.
extern "C" unsigned char *g_resultData;

// Current state of world.
extern "C" unsigned char *g_data;

// Current width of world.
extern "C" size_t g_worldWidth;

// Current height of world.
extern "C" size_t g_worldHeight;

// Current data length (product of width and height)
size_t g_dataLength=0;  // g_worldWidth * g_worldHeight


//allocate ghost row here
extern "C" unsigned char *my_first_row; // first row of the current rank
extern "C" unsigned char *my_last_row; // last row of the current rank

extern "C" unsigned char *previous_last_row; //  last row of the previous rank
extern "C" unsigned char *next_first_row; // first row of the next rank

extern "C" void init_Ghost_rows()
{
    // ghost row size is the same as world width because its just one row
    hipMallocManaged(&my_first_row, (g_worldWidth * sizeof(unsigned char)));
    hipMallocManaged(&my_last_row, (g_worldWidth * sizeof(unsigned char)));

    // Initialize values by assigning values from g_data
    int i;
    for(i=0; i< g_worldWidth; i++)
        my_first_row[i] = g_data[i];

    size_t x = g_worldWidth*(g_worldHeight - 1);
    for(i = 0; i<g_worldWidth; i++){
        my_last_row[i] = g_data[x];
        x++;
    }

    // ghost row size is the same as world width because its just one row
    hipMallocManaged(&previous_last_row, (g_worldWidth * sizeof(unsigned char))); // no initial value, will be updated via Irecv
    hipMallocManaged(&next_first_row, (g_worldWidth * sizeof(unsigned char))); // no initial value, will be updated via Irecv

}

//pattern 0
static inline void gol_initAllZeros( size_t worldWidth, size_t worldHeight, int myRank, int numRank )
{
    g_worldWidth = worldWidth;
    g_worldHeight = worldHeight;
    g_dataLength = g_worldWidth * g_worldHeight;

    const size_t sz = size_t(g_dataLength) * sizeof(unsigned char);

    hipMallocManaged( &g_data, (g_dataLength * sizeof(unsigned char)));
    hipMemset(g_data, 0, sz);

    hipMallocManaged( &g_resultData, (g_dataLength * sizeof(unsigned char)));
    hipMemset(g_resultData, 0, sz);
}

//pattern 1
static inline void gol_initAllOnes( size_t worldWidth, size_t worldHeight, int myRank, int numRank )
{
    int i;

    g_worldWidth = worldWidth;
    g_worldHeight = worldHeight;
    g_dataLength = g_worldWidth * g_worldHeight;

    const size_t sz = size_t(g_dataLength) * sizeof(unsigned char);

    hipMallocManaged( &g_data, (g_dataLength * sizeof(unsigned char)));

    // set all rows of world to true
    for( i = 0; i < g_dataLength; i++)
    {
        g_data[i] = 1;
    }

   hipMallocManaged( &g_resultData, (g_dataLength * sizeof(unsigned char)));
   hipMemset(g_resultData, 0, sz);
}

//pattern 2
static inline void gol_initOnesInMiddle( size_t worldWidth, size_t worldHeight, int myRank, int numRank )
{
    int i;

    g_worldWidth = worldWidth;
    g_worldHeight = worldHeight;
    g_dataLength = g_worldWidth * g_worldHeight;

    const size_t sz = size_t(g_dataLength) * sizeof(unsigned char);

    hipMallocManaged( &g_data, (g_dataLength * sizeof(unsigned char)));
    hipMemset(g_data, 0, sz);

    int row_offset = (worldHeight - 1)*worldHeight;
    int col_offset = row_offset + 127;

    for( i = col_offset; i < col_offset+10; i++)
    {
        g_data[i] = 1;  
    }

    //allocate memory for resultData
    hipMallocManaged( &g_resultData, (g_dataLength * sizeof(unsigned char)));
    // fill in resultData with zeroes
    hipMemset(g_resultData, 0, sz);
}

//pattern 3
static inline void gol_initOnesAtCorners( size_t worldWidth, size_t worldHeight, int myRank, int numRank )
{
    g_worldWidth = worldWidth;
    g_worldHeight = worldHeight;
    g_dataLength = g_worldWidth * g_worldHeight;

    const size_t sz = size_t(g_dataLength) * sizeof(unsigned char);

    hipMallocManaged( &g_data, (g_dataLength * sizeof(unsigned char)));
    hipMemset(g_data, 0, sz);
    
    if(myRank == 0){
        g_data[0] = 1; // upper left
        g_data[worldWidth-1]=1; // upper right
    }
    
    else if(myRank == numRank - 1){
        g_data[(worldHeight * (worldWidth-1))]=1; // lower left
        g_data[(worldHeight * (worldWidth-1)) + worldWidth-1]=1; // lower right
    }
    
    
    hipMallocManaged( &g_resultData, (g_dataLength * sizeof(unsigned char)));
    hipMemset(g_resultData, 0, sz);
}

//pattern 4
static inline void gol_initSpinnerAtCorner( size_t worldWidth, size_t worldHeight, int myRank, int numRank )
{
    g_worldWidth = worldWidth;
    g_worldHeight = worldHeight;
    g_dataLength = g_worldWidth * g_worldHeight;

    const size_t sz = size_t(g_dataLength) * sizeof(unsigned char);

    hipMallocManaged( &g_data, (g_dataLength * sizeof(unsigned char)));
    hipMemset(g_data, 0, sz);
    
    if(myRank == 0){
        g_data[0] = 1; // upper left
        g_data[1] = 1; // upper left +1
        g_data[worldWidth-1]=1; // upper right

    }

    hipMallocManaged( &g_resultData, (g_dataLength * sizeof(unsigned char)));
    hipMemset(g_resultData, 0, sz);

}

// function that swaps the result words and original world back to back
extern "C" static inline void gol_swap( unsigned char **pA, unsigned char **pB)
{
    // The type of the temporary variable is the same as g_data and g_resultData
    // Because the **pA and **pB are pointers to g_data and g_resultData
    unsigned char *temp = NULL;

    //swap the variables

    temp = *pA;
    *pA = *pB;
    *pB = temp;

}

// the kernel function that updates the cells
__global__ void gol_kernel(const unsigned char* d_data,
unsigned int worldWidth,
unsigned int worldHeight,
unsigned char* d_resultData,
unsigned char* previous_last_row,
unsigned char* next_first_row){

    int index;
    // calculate index and iterate over
    for(index = blockIdx.x *blockDim.x + threadIdx.x; index < worldWidth*worldHeight; index += blockDim.x * gridDim.x)
    {
        size_t x, y;

        // calculate a cell position in 2D using index
        y = (int) index/worldWidth;
        x = index - y*worldWidth;

        
        // calculating values for x0, x1 and x2 in 1-D
        size_t x0, x1, x2;
        x1 = x;
        x0 = (x + worldWidth - 1) % worldWidth;
        x2 = (x + 1) % worldWidth;


        // calculating values for y0, y1 and y2 in 1-D
        size_t y0, y1, y2;

        y0=(y-1)*worldWidth;
        y1=y*worldWidth;
        y2=(y+1)*worldWidth;

        
        int alive_neighbors = 0;
        printf("BEFORE AND");
        if(y1 < worldWidth) // if the current row is the first row then take y0 values from the ghost row (previous rank's last row)
            alive_neighbors = (unsigned int)previous_last_row[x0] + (unsigned int)d_data[x0+y1] + (unsigned int)d_data[x0+y2] + 
                            (unsigned int)previous_last_row[x1] + (unsigned int)d_data[x1+y2] + 
                            (unsigned int)previous_last_row[x2] + (unsigned int)d_data[x2+y1] + (unsigned int)d_data[x2+y2];
        else if(y1 >= worldWidth*(worldHeight - 1)) // if the current row is the last row then take y0 values from the ghost row (next rank's first row)
            alive_neighbors = (unsigned int)d_data[x0+y0] + (unsigned int)d_data[x0+y1] + (unsigned int)next_first_row[x0] + 
                        (unsigned int)d_data[x1+y0] + (unsigned int)next_first_row[x1] + 
                        (unsigned int)d_data[x2+y0] + (unsigned int)d_data[x2+y1] + (unsigned int)next_first_row[x2];        
        else // calculate as usual
            alive_neighbors = (unsigned int)d_data[x0+y0] + (unsigned int)d_data[x0+y1] + 
                            (unsigned int)d_data[x0+y2] + (unsigned int)d_data[x1+y0] + 
                            (unsigned int)d_data[x1+y2] + (unsigned int)d_data[x2+y0] + (unsigned int)d_data[x2+y1] + (unsigned int)d_data[x2+y2];

        // If a cell is alive, there are three possible scenarios
        // Either it dies due to overpopulation or underppopulation or it remains alive
        if(d_data[y1 + x] == 1){
            if(alive_neighbors<2)
                d_resultData[y1 + x] = 0;
            else if (alive_neighbors ==2 || alive_neighbors == 3)
                d_resultData[y1 + x] = 1;
            else if (alive_neighbors>3){
                d_resultData[y1+x] = 0;

            }
        }
        // If a cell is dead, there are two scenarios
        // either it is alive again or remains dead
        else if(d_data[y1+x] == 0)
        {
            if(alive_neighbors == 3)
                d_resultData[y1+x] = 1;
            else
                d_resultData[y1+x] = 0;
    }


    }

}
// this function calls the kernel
extern "C" bool gol_kernelLaunch(unsigned char** d_data,
unsigned char** d_resultData,
size_t worldWidth,
size_t worldHeight,
size_t iterationsCount,
ushort threadsCount){

    // number of block is calculated
    size_t blocksCount = (size_t)((worldWidth*worldHeight)/threadsCount);
    
    // calling kernel
    gol_kernel<<<blocksCount, threadsCount>>>(*d_data, worldWidth, worldHeight, *d_resultData, previous_last_row, next_first_row);
    // synchronization ensures the end of execution for all kernels before swapping 
    hipDeviceSynchronize();
    // swap the worlds
    gol_swap(d_data, d_resultData);
    

    return true;

}
// initialize the world using the pattern
extern "C" void gol_initMaster( unsigned int pattern, size_t worldWidth, size_t worldHeight, int myRank, int numRank )
{
    int cudaDeviceCount;
    hipError_t cE;
    if( (cE = hipGetDeviceCount( &cudaDeviceCount)) != hipSuccess )
    {
        printf(" Unable to determine cuda device count, error is %d, count is %d\n",
        cE, cudaDeviceCount );
        exit(-1);
    }
    if( (cE = hipSetDevice( myRank % cudaDeviceCount )) != hipSuccess )
    {
        printf(" Unable to have rank %d set to cuda device %d, error is %d \n",
        myRank, (myRank % cudaDeviceCount), cE);
        exit(-1);
    }
    switch(pattern)
    {
    case 0:
        gol_initAllZeros( worldWidth, worldHeight, myRank, numRank );
        break;

    case 1:
        gol_initAllOnes( worldWidth, worldHeight, myRank, numRank );
        break;

    case 2:
        gol_initOnesInMiddle( worldWidth, worldHeight, myRank, numRank );
        break;

    case 3:
        gol_initOnesAtCorners( worldWidth, worldHeight, myRank, numRank );
        break;

    case 4:
        gol_initSpinnerAtCorner( worldWidth, worldHeight, myRank, numRank );
        break;

    default:
        printf("Pattern %u has not been implemented \n", pattern);
        exit(-1);
    }
}





extern "C" void cuda_finalize()
{
    //free up memory
    hipFree(g_data);
    hipFree(g_resultData);
}


